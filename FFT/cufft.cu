
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
 
// Declared extern "C" to disable C++ name mangling
extern "C" void launchCUFFT(float *d_data, int n)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, (hipfftComplex*)d_data, (hipfftComplex*)d_data,HIPFFT_FORWARD);
    hipfftDestroy(plan);
}
