
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
 
// Note the trailing underscore and that scalar arguments are passed by reference for Fortran compatibility
extern "C" void launchcufft_(float *d_data, int *length)
{
    int n = *length;
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, (hipfftComplex*)d_data, (hipfftComplex*)d_data,HIPFFT_FORWARD);
    hipfftDestroy(plan);
}
