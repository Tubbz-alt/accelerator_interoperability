#include <hipfft/hipfft.h>

// Note the trailing underscore and that scalar arguments are passed by reference for Fortran compatibility
extern "C" void launchcufft_(float *d_data, int *length, void *stream)
{
    int n = *length;
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    hipfftSetStream(plan, stream);
    hipfftExecC2C(plan, (hipfftComplex*)d_data, (hipfftComplex*)d_data,HIPFFT_FORWARD);
    hipfftDestroy(plan);
}
